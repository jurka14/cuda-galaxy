#include "hip/hip_runtime.h"
#define MAX_BLOCK 64


__global__ void compute(sGalaxy A, sGalaxy B, int n, float* res)
{
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int bx = blockIdx.x;
    int tx = threadIdx.x;
    int tile_size = blockDim.x;

    float axi = A.x[i];
    float ayi = A.y[i];
    float azi = A.z[i];
    float bxi = B.x[i];
    float byi = B.y[i];
    float bzi = B.z[i];
    
    __shared__ float Ax[MAX_BLOCK];
    __shared__ float Ay[MAX_BLOCK];
    __shared__ float Az[MAX_BLOCK];
    __shared__ float Bx[MAX_BLOCK];
    __shared__ float By[MAX_BLOCK];
    __shared__ float Bz[MAX_BLOCK];
    
    float tmp = 0.0f;

    for (int b = bx; b < n/tile_size + 1; b++)
    {
        if (i < n)
        {
            Ax[tx] = A.x[b*tile_size + tx];
            Ay[tx] = A.y[b*tile_size + tx];
            Az[tx] = A.z[b*tile_size + tx];
            Bx[tx] = B.x[b*tile_size + tx];
            By[tx] = B.y[b*tile_size + tx];
            Bz[tx] = B.z[b*tile_size + tx];
        }
        
        
        __syncthreads();
        
        for (int j = 0; j < tile_size; j++)
        {
            if (b*tile_size+j > i && b*tile_size+j < n)
            {
                float da = sqrt((axi-Ax[j])*(axi-Ax[j])
                    + (ayi-Ay[j])*(ayi-Ay[j])
                    + (azi-Az[j])*(azi-Az[j]));
                float db = sqrt((bxi-Bx[j])*(bxi-Bx[j])
                    + (byi-By[j])*(byi-By[j])
                    + (bzi-Bz[j])*(bzi-Bz[j]));
                tmp += (da-db) * (da-db);
            }
        }
        __syncthreads();
    }

    if(i < n-1)
    {
        res[i] = tmp;
    }

}

float solveGPU(sGalaxy A, sGalaxy B, int n) 
{
    size_t size = n * sizeof(float);
    float* h_res = (float*)malloc(size);
    float* d_res;
    hipMalloc(&d_res, size);

    int block = 32;
    
    compute<<<n/block+1, block>>>(A, B, n, d_res);

    hipMemcpy(h_res, d_res, size, hipMemcpyDeviceToHost);
    

    float res = 0.0f;
    for (int i = 0; i < n-1; i++) 
    {
        res += h_res[i];
    }

    res = sqrt(1/((float)n*((float)n-1)) * res);
    
    hipFree(d_res);
    free(h_res);

    return res;
}